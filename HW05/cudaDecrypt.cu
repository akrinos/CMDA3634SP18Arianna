#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE * key; 
  key = fopen("public_key.txt", "r");
  char * currLine = NULL; size_t length = 0;
  getline(&currLine, &length, key);
  n = atoi(currLine); currLine = NULL;
  getline(&currLine, &length, key);
  p = atoi(currLine); currLine = NULL;
  getline(&currLine, &length, key);
  g = atoi(currLine), currLine = NULL;
  getline(&currLine, &length, key);
  h = atoi(currLine); fclose(key); 

  FILE * messge;
  messge = fopen("message.txt", "r");
  currLine = NULL; length = 0;
  getline(&currLine, &length, key);
  int numEnt = atoi(currLine); currLine = NULL;
  unsigned int * m = malloc(numEnt * sizeof(unsigned int));
  unsigned int * a = malloc(numEnt * sizeof(unsigned int));
  for (int i = 0; i < numEnt; i++) {
	getline(&currLine, &length, key);
	m[i] = atoi(strtok(currLine, " "));
	a[i] = atoi(strtok(NULL, " ")); 
//	printf("we've got %d and %d\n", m[i],a[i]);
	currLine = NULL; 
  } 

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */
  // Feed in [m] and [a] to the decrypt function 
  Nints = numEnt;
  unsigned int Nchars = numEnt * ((n - 1) / 8);
  ElGamalDecrypt(m, a, Nints, p, x);
  char * final = malloc(Nchars * sizeof(char));
  convertZToString(m, Nints, final, Nchars);
  // printf("%s\n", final); 
  return 0;
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
