#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <stdbool.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//compute a*b mod p safely
__device__ unsigned int modprodCuda(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int modExpCuda(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprodCuda(aExpb, z, p);
    z = modprodCuda(z, z, p);
    b /= 2;
  }
  return aExpb;
}

__global__ void findTheX(volatile unsigned int *xres, unsigned int p, unsigned int h, unsigned int g) {
	int threadid = threadIdx.x; //thread number
	int blockid = blockIdx.x; //block number
	int Nblock = blockDim.x;  //number of threads in a block
	
	unsigned int id = threadid + blockid*Nblock;
	printf("hello from id %u \n", id);
	if ((id < p) && modExpCuda(g,id+1,p)==h) {
		*xres = id+1;
		// bug: xres is getting set to number of blocks always 
		//__syncthreads();
	}
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE * key; 
  key = fopen("public_key.txt", "r");
  char * currLine = NULL; size_t length = 0;
  getline(&currLine, &length, key);
  n = atoi(currLine); currLine = NULL;
  getline(&currLine, &length, key);
  p = atoi(currLine); currLine = NULL;
  getline(&currLine, &length, key);
  g = atoi(currLine), currLine = NULL;
  getline(&currLine, &length, key);
  h = atoi(currLine); fclose(key); 

  FILE * messge;
  messge = fopen("message.txt", "r");
  currLine = NULL; length = 0;
  getline(&currLine, &length, key);
  int numEnt = atoi(currLine); currLine = NULL;
  unsigned int * m = (unsigned int *) malloc(numEnt * sizeof(unsigned int));
  unsigned int * a = (unsigned int *) malloc(numEnt * sizeof(unsigned int));
  for (int i = 0; i < numEnt; i++) {
	getline(&currLine, &length, key);
	m[i] = atoi(strtok(currLine, " "));
	a[i] = atoi(strtok(NULL, " ")); 
//	printf("we've got %d and %d\n", m[i],a[i]);
	currLine = NULL; 
  } 

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    // Comment the following lines if you don't want to do the full loop
    
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found by loop! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double ending = clock();

    double totalTime = (ending-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
    
    double starting = clock();
    unsigned int *x_res;
    hipMalloc(&x_res, 1*sizeof(unsigned int));
    hipMemcpy(x_res, 0, 1 * sizeof(unsigned int), hipMemcpyHostToDevice);
    // the number of blocks we have corresponds to independent
    // executions in parallel - our design, we are skipping forward
    // by multiples so that 
    //dim3 B(128, 1, 1);
    //dim3 G((N + 128 - 1) / 128, 1, 1);

    int Nthreads = 128;
    int Nblocks = (p + Nthreads - 1) / Nthreads;
    // p, g, and h are just constants
    printf("prevals: %d, %d, %d\n", *x_res, p, h); 
    findTheX <<<Nblocks, Nthreads>>> (x_res, p, h, g);
    hipDeviceSynchronize();
    // Hopefully by this point we've found the x 
    unsigned int *result;
    hipMemcpy(result, x_res, 1*sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("We found the secret key as %d\n", result);
    x = *result;
    double endTime = clock();

    double totalTimec = (endTime-starting)/CLOCKS_PER_SEC;
    double workc = (double) p;
    double throughputc = workc/totalTimec;

    printf("Searching all keys using cuda took %g seconds, throughput was %g values tested per second.\n", totalTimec, throughputc);
  }

  /* Q3 After finding the secret key, decrypt the message */
  // Feed in [m] and [a] to the decrypt function 
  Nints = numEnt;
  unsigned int Nchars = numEnt * ((n - 1) / 8);
  ElGamalDecrypt(m, a, Nints, p, x);
  unsigned char * final = (unsigned char *) malloc(Nchars * sizeof(char));
  convertZToString(m, Nints, final, Nchars);
  printf("%s\n", final); 
  return 0;
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
}
