#include "hip/hip_runtime.h"
/* 

To compile:

   nvcc -O3 -o mandelbrot mandelbrot.c png_util.c -I. -lpng -lm -fopenmp

Or just type:

   module load gcc
   make

To create an image with 4096 x 4096 pixels (last argument will be used to set number of threads):

    ./mandelbrot 4096 4096 1

*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "png_util.h"

// Q2a: add include for CUDA header file here:
#include "hip/hip_runtime.h"

#define MXITER 1000

typedef struct {
  
  double r;
  double i;
  
}complex_t;

// return iterations before z leaves mandelbrot set for given c
__device__ int testpoint(complex_t c){
  
  int iter;

  complex_t z;
  double temp;
  
  z = c;
  
  for(iter=0; iter<MXITER; iter++){
    
    temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i)>4.0){
      return iter;
    }
  }
  
  
  return iter;
  
}

// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array

// Q2c: transform this function into a CUDA kernel
__global__ void  kernelMandelbrot(int Nre, int Nim, complex_t cmin, complex_t cmax, float *count){ 
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int bSizex = blockDim.x;
  int bSizey = blockDim.y;

//  int n,m;

  complex_t c;

  double dr = (cmax.r-cmin.r)/(Nre-1);
  double di = (cmax.i-cmin.i)/(Nim-1);;

  int i = tx + bx * bSizex; // x coord
  int j = ty + by * bSizey; // y coord
  
  c.r = cmin.r + dr * i;
  c.i = cmin.i + di + j;
  //c.r = cmin.r + ty * i;
  //c.i = cmin.i + tx * j;
  count[i + j * Nre] = testpoint(c);
//  for(n=0;n<Nim;++n){
//   for(m=0;m<Nre;++m){
//      c.r = cmin.r + dr*m; // the m number is the real index 
//     c.i = cmin.i + di*n; // the n number is the imaginary index
      
//      count[m+n*Nre] = testpoint(c);
      
//    }
 // }

}

int main(int argc, char **argv){

  // to create a 4096x4096 pixel image [ last argument is placeholder for number of threads ] 
  // usage: ./mandelbrot 4096 4096 1  
  printf("here we go\n");  

  int Nre = atoi(argv[1]);
  int Nim = atoi(argv[2]);
  printf("Nre and Nim are %d and %d\n", Nre, Nim);
  int Nthreads = Nre * Nim; //atoi(argv[3]);
  int N = Nre * Nim;

  // Q2b: set the number of threads per block and the number of blocks here:
  float *fltArray;
  hipMalloc(&fltArray, Nre*Nim*sizeof(float));

  int Nblocks = (int) (N + Nthreads - 1) / (Nthreads); 
  int Bx, By, Gx, Gy; 
  Bx = Nthreads;
  By = Nthreads;
  Gx = Nblocks;
  Gy = Nblocks;

  dim3 block(Bx, By, 1); // Bx * By threads in thread-block
  dim3 grid(Gx, Gy, 1); // Gx * Gy grid of thread-blocks 

  // storage for the iteration counts
  float *count = (float*) malloc(Nre*Nim*sizeof(float));

  // Parameters for a bounding box for "c" that generates an interesting image
  const float centRe = -.759856, centIm= .125547;
  const float diam  = 0.151579;

  complex_t cmin; 
  complex_t cmax;

  cmin.r = centRe - 0.5*diam;
  cmax.r = centRe + 0.5*diam;
  cmin.i = centIm - 0.5*diam;
  cmax.i = centIm + 0.5*diam;

  clock_t start = clock(); //start time in CPU cycles

  // compute mandelbrot set
  kernelMandelbrot<<<Nblocks , Nthreads >>>(Nre, Nim, cmin, cmax, count); 
  
  clock_t end = clock(); //start time in CPU cycles
  
  // print elapsed time
  printf("elapsed = %f\n", ((double)(end-start))/CLOCKS_PER_SEC);
  hipMemcpy(fltArray, count, Nthreads * sizeof(float), hipMemcpyDeviceToHost);

  // output mandelbrot to png format image
  //char Nre_str = malloc(sizeof(char)*(int)log10(Nre));
  //sprintf(&Nre_str, "%d", Nre);
  char randomLet = 'A' + (random() % 26);
  char* fileName = "mandelbrot"; 
  strcat(fileName, &randomLet);
  strcat(fileName, ".png");
  FILE *fp = fopen(fileName, "w");

  printf("Printing mandelbrot.png...");
  write_hot_png(fp, Nre, Nim, count, 0, 80);
  printf("done.\n");

  free(count);

  exit(0);
  return 0;
}  
